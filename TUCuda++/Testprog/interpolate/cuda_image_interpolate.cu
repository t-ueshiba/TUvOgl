#include "hip/hip_runtime.h"
/*
 * $Id: cuda_image_interpolate.cu,v 1.2 2012-08-30 12:19:21 ueshiba Exp $
 */
#include "TU/Image++.h"
#include "TU/Profiler.h"
#include "TU/cuda/Array++.h"
#include "TU/cuda/chrono.h"

namespace TU
{
namespace cuda
{
template <class T> __device__ T
interpolate_pixel(T s0, T s1, float r0, float r1)
{
    return s0 * r0 + s1 * r1;
}

#if 0
template <> __device__ RGBA
interpolate_pixel(RGBA s0, RGBA s1, float r0, float r1)
{
    RGBA	val;
    val.r = s0.r * r0 + s1.r * r1;
    val.g = s0.g * r0 + s1.g * r1;
    val.b = s0.b * r0 + s1.b * r1;
    
    return val;
}
#endif
    
template <class T> __global__ void
interpolate_kernel(const T* src0, const T* src1, T* dst,
		   u_int stride, float ratio)
{
    const u_int	xy = (blockIdx.y * blockDim.y + threadIdx.y) * stride
		   +  blockIdx.x * blockDim.x + threadIdx.x;

    dst[xy] = interpolate_pixel(src0[xy], src1[xy], ratio, 1.0f - ratio);
}
    
template <class T> void
interpolate(const Array2<T>& d_image0,
	    const Array2<T>& d_image1, Array2<T>& d_image2)
{
    using namespace	std;

    d_image2.resize(d_image0.nrow(), d_image0.ncol());
    
  // timer
    Profiler<clock>	cuProfiler(1);

  // setup execution parameters
    dim3  threads(16, 16, 1);
    dim3  blocks(d_image0.ncol()/threads.x, d_image0.nrow()/threads.y, 1);
    cerr << blocks.x << 'x' << blocks.y << " blocks..." << endl;
    
  // execute the kernel
    cerr << "Let's go!" << endl;
    for (int i = 0; i < 1000; ++i)
    {
	cuProfiler.start(0);
	interpolate_kernel<<<blocks, threads>>>(d_image0.data().get(),
						d_image1.data().get(),
						d_image2.data().get(),
						d_image2.stride(), 0.5f);
	cuProfiler.stop();
	cuProfiler.nextFrame();
    }
    cerr << "Returned!" << endl;
    cuProfiler.print(cerr);
}

template void	interpolate(const Array2<u_char>& d_image0,
			    const Array2<u_char>& d_image1,
				  Array2<u_char>& d_image2)	;
  /*
template void	interpolate(const Array2<RGBA>&   d_image0,
			    const Array2<RGBA>&   d_image1,
				  Array2<RGBA>&   d_image2)	;
template void	interpolate(const Array2<float4>& d_image0,
			    const Array2<float4>& d_image1,
				  Array2<float4>& d_image2)	;
  */
}
}
