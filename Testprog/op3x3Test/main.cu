#include "hip/hip_runtime.h"
/*
 *  $Id: main.cu,v 1.1 2012-08-30 00:13:51 ueshiba Exp $
 */
#include "TU/Image++.h"
#include "TU/Profiler.h"
#include "TU/cuda/Array++.h"
#include "TU/cuda/algorithm.h"
#include "TU/cuda/functional.h"
#include "TU/cuda/chrono.h"

//#define OP	cuda::det3x3
//#define OP	cuda::laplacian3x3
//#define OP	cuda::sobelAbs3x3
#define OP	cuda::maximal3x3
//#define OP	cuda::minimal3x3

/************************************************************************
*  Global fucntions							*
************************************************************************/
int
main(int argc, char *argv[])
{
    using namespace	std;
    using namespace	TU;

  //using in_t	= u_char;
  //using out_t	= u_char;
    using in_t	= float;
    using out_t	= float;
    
    try
    {
	Image<in_t>	in;
	in.restore(cin);				// 原画像を読み込む
	in.save(cout);					// 原画像をセーブ

      // GPUによって計算する．
	cuda::Array2<in_t>	in_d(in);
	cuda::Array2<out_t>	out_d(in.nrow(), in.ncol());
	cuda::op3x3(in_d.cbegin(), in_d.cend(), out_d.begin(), OP<in_t>());
	hipDeviceSynchronize();

	Profiler<cuda::clock>	cuProfiler(1);
	constexpr size_t	NITER = 1000;
	for (size_t n = 0; n < NITER; ++n)		// フィルタリング
	{
	    cuProfiler.start(0);
	    cuda::op3x3(in_d.cbegin(), in_d.cend(), out_d.begin(), OP<in_t>());
	    cuProfiler.nextFrame();
	}
	cuProfiler.print(std::cerr);
	
	Image<out_t>	out(out_d);
	out.save(cout);					// 結果画像をセーブ

      // CPUによって計算する．
	Profiler<>	profiler(1);
	Image<out_t>	outGold;
	for (size_t n = 0; n < 10; ++n)
	{
	    outGold = in;
	    profiler.start(0);
	    op3x3(outGold.begin(), outGold.end(), OP<in_t>());
	    profiler.nextFrame();
	}
	profiler.print(cerr);
	outGold.save(cout);

      // 結果を比較する．
	const int	V = 160;
	for (size_t u = 0; u < out.width(); ++u)
	    cerr << ' ' << (out[V][u] - outGold[V][u]);
	cerr <<  endl;
    }
    catch (exception& err)
    {
	cerr << err.what() << endl;
	return 1;
    }

    return 0;
}
